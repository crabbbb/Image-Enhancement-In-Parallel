#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_complex.h>
#include <cmath>


using namespace std;

// CUDA Kernel to compute Gaussian High-Pass Filter values
__global__ void computeHighPassKernel(double* H, int width, int height, double cutoff_frequency) {
    int u = blockIdx.y * blockDim.y + threadIdx.y;
    int v = blockIdx.x * blockDim.x + threadIdx.x;

    if (u < height && v < width) {
        double D_uv = sqrt(pow(u - height / 2.0, 2) + pow(v - width / 2.0, 2));
        H[u * width + v] = 1.0 - exp(-pow(D_uv, 2) / (2 * pow(cutoff_frequency, 2)));
    }
}

// CUDA Kernel to apply the filter to the frequency-domain data
__global__ void applyFilterKernel(hipDoubleComplex* F_shifted, double* H, hipDoubleComplex* G, int width, int height) {
    int u = blockIdx.y * blockDim.y + threadIdx.y;
    int v = blockIdx.x * blockDim.x + threadIdx.x;

    if (u < height && v < width) {
        int idx = u * width + v;
        hipDoubleComplex value = make_hipDoubleComplex(H[idx] * hipCreal(F_shifted[idx]), H[idx] * hipCimag(F_shifted[idx]));
        G[idx] = value;
    }
}

// Host function to perform the Gaussian High-Pass Filter
hipDoubleComplex** gaussianHighPassFilterCUDA(hipDoubleComplex** F_shifted, int width, int height, double cutoff_frequency) {
    // Allocate memory for the filter on the device
    double* d_H;
    hipMalloc(&d_H, width * height * sizeof(double));

    // Allocate memory for the input and output frequency-domain data
    hipDoubleComplex* d_F_shifted;
    hipDoubleComplex* d_G;
    hipMalloc(&d_F_shifted, width * height * sizeof(hipDoubleComplex));
    hipMalloc(&d_G, width * height * sizeof(hipDoubleComplex));

    // Flatten the 2D F_shifted array for CUDA
    hipDoubleComplex* h_F_shifted = new hipDoubleComplex[width * height];
    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            h_F_shifted[i * width + j] = F_shifted[i][j];
        }
    }
    hipMemcpy(d_F_shifted, h_F_shifted, width * height * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // Define CUDA grid and block sizes
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // Compute the filter values
    computeHighPassKernel <<< gridSize, blockSize >>> (d_H, width, height, cutoff_frequency);
    hipDeviceSynchronize();

    // Apply the filter
    applyFilterKernel <<< gridSize, blockSize >>> (d_F_shifted, d_H, d_G, width, height);
    hipDeviceSynchronize();

    // Copy the result back to the host
    hipDoubleComplex* h_G = new hipDoubleComplex[width * height];
    hipMemcpy(h_G, d_G, width * height * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Convert the flattened result back to a 2D array
    hipDoubleComplex** G = new hipDoubleComplex * [height];
    for (int i = 0; i < height; ++i) {
        G[i] = new hipDoubleComplex[width];
        for (int j = 0; j < width; ++j) {
            G[i][j] = h_G[i * width + j];
        }
    }

    // Cleanup
    delete[] h_F_shifted;
    delete[] h_G;
    hipFree(d_H);
    hipFree(d_F_shifted);
    hipFree(d_G);

    return G;
}

// Test function for CUDA Gaussian High-Pass Filter
bool testGaussianHighPassFilterCUDA() {
    // Image dimensions
    const int width = 4;
    const int height = 4;

    // Cutoff frequency
    double cutoff_frequency = 2.0;

    // Simulated frequency-domain data (4x4 matrix)
    hipDoubleComplex** F_shifted = new hipDoubleComplex * [height];
    for (int i = 0; i < height; ++i) {
        F_shifted[i] = new hipDoubleComplex[width];
    }

    int value = 1;
    for (int u = 0; u < height; ++u) {
        for (int v = 0; v < width; ++v) {
            F_shifted[u][v] = make_hipDoubleComplex(value, value);
            ++value;
        }
    }

    // Call the CUDA Gaussian High-Pass Filter
    hipDoubleComplex** G = gaussianHighPassFilterCUDA(F_shifted, width, height, cutoff_frequency);

    // Expected output (manually calculated for this example)
    bool test_passed = true;
    for (int u = 0; u < height; ++u) {
        for (int v = 0; v < width; ++v) {
            double D_uv = sqrt(pow(u - height / 2.0, 2) + pow(v - width / 2.0, 2));
            double H_uv = 1.0 - exp(-pow(D_uv, 2) / (2 * pow(cutoff_frequency, 2)));
            hipDoubleComplex expected = make_hipDoubleComplex(
                hipCreal(F_shifted[u][v]) * H_uv,
                hipCimag(F_shifted[u][v]) * H_uv
            );

            // Allow for small floating-point differences
            if (abs(hipCreal(G[u][v]) - hipCreal(expected)) > 1e-6 ||
                abs(hipCimag(G[u][v]) - hipCimag(expected)) > 1e-6) {
                test_passed = false;
                cout << "Mismatch at (" << u << ", " << v << "): "
                    << "Expected (" << hipCreal(expected) << ", " << hipCimag(expected)
                    << "), Got (" << hipCreal(G[u][v]) << ", " << hipCimag(G[u][v]) << ")" << endl;
            }
        }
    }

    // Cleanup dynamically allocated arrays
    for (int i = 0; i < height; ++i) {
        delete[] F_shifted[i];
        delete[] G[i];
    }
    delete[] F_shifted;
    delete[] G;

    return test_passed;
}

