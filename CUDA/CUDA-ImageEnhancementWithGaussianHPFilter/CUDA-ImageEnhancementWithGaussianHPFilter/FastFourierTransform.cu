#include "hip/hip_runtime.h"
/************************************************************************
 * Manual 2D FFT in CUDA using Cooley-Tukey - Supports Rectangular Images
 * ----------------------------------------------------------------------
 * - Accepts/returns hipDoubleComplex** at the host side
 * - Internally flattens data into a 1D GPU buffer for batched processing
 * - Row transform:  length = width, batchCount = height
 * - Transpose       (height x width) -> (width x height)
 * - Column transform: length = height, batchCount = width
 * - Transpose back
 *
 * For inverse transform, each 1D pass scales by (1 / length), so total
 * scaling is (1 / (width*height)).
 ************************************************************************/

#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_complex.h>
#include <iostream>
#include <cmath>
#include <cstdlib>   // for std::exit
#include <cstring>   // for std::memcpy
#include "Utils.hpp"

using namespace std;

#ifndef M_PI
#define M_PI 3.14159265358979323846
#endif

 // ---------------------------------------------------------------------
 // Simple CUDA error-checking macro
 // ---------------------------------------------------------------------
#define CHECK_CUDA_ERR(msg) do {                                 \
    hipError_t err = hipGetLastError();                        \
    if (err != hipSuccess) {                                    \
        std::cerr << msg << " (error " << err << "): "           \
                  << hipGetErrorString(err) << std::endl;       \
        std::exit(EXIT_FAILURE);                                 \
    }                                                            \
} while(0)

// =====================================================================
// 1) Kernels for batched 1D FFT (Cooley�Tukey) in "row-major" slices
// =====================================================================

// (a) Bit-reversal permutation for each "row" of length = `length`
__global__ void BitReverseRowKernel(
    hipDoubleComplex* d_data,
    int length,       // how many elements per row transform
    int batchCount,   // how many rows/batches
    int log2Len       // log2(length) (number of bits in each element)
)
{
    int batch = blockIdx.y;  // which row/batch
    if (batch >= batchCount) return;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        // bit-reverse 'tid' in the range 0 to length - 1
        int x = tid;
        int r = 0;
        // process log2Len bits, after the loop, reversed bits of the current index is assigned to r.
        for (int i = 0; i < log2Len; i++) {
            r = (r << 1) | (x & 1);
            x >>= 1;
        }
        if (r > tid) {
            int rowOffset = batch * length;
            // swap
            hipDoubleComplex tmp = d_data[rowOffset + tid];
            d_data[rowOffset + tid] = d_data[rowOffset + r];
            d_data[rowOffset + r] = tmp;
        }
    }
}

// (b) One pass of the Cooley�Tukey butterfly for each row
__global__ void CooleyTukeyRowKernel(
    hipDoubleComplex* d_data,
    int length,
    int batchCount,
    int halfSize,
    int sign // -1 => forward, +1 => inverse
)
{
    int batch = blockIdx.y;
    if (batch >= batchCount) return;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalButterflies = length / 2; // # of butterfly pairs per row

    if (tid < totalButterflies) {
        int blockIndex = tid / halfSize;
        int t = tid % halfSize;
        int butterflySz = 2 * halfSize;

        int rowOffset = batch * length;
        int evenIndex = rowOffset + blockIndex * butterflySz + t;
        int oddIndex = evenIndex + halfSize;

        hipDoubleComplex even = d_data[evenIndex];
        hipDoubleComplex odd = d_data[oddIndex];

        double angle = sign * 2.0 * M_PI * t / (double)butterflySz;
        hipDoubleComplex w = make_hipDoubleComplex(cos(angle), sin(angle));

        hipDoubleComplex product = hipCmul(w, odd);
        d_data[evenIndex] = hipCadd(even, product);
        d_data[oddIndex] = hipCsub(even, product);
    }
}

// (c) Scale kernel: multiply each element in row by (1 / length) if doing inverse
__global__ void ScaleRowKernel(
    hipDoubleComplex* d_data,
    int length,
    int batchCount,
    double scale
)
{
    int batch = blockIdx.y;
    if (batch >= batchCount) return;

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < length) {
        int idx = batch * length + tid;
        double re = hipCreal(d_data[idx]);
        double im = hipCimag(d_data[idx]);
        d_data[idx] = make_hipDoubleComplex(scale * re, scale * im);
    }
}

// =====================================================================
// 2) Kernel to transpose a (height x width) buffer into (width x height)
//    d_out[col * height + row] = d_in[row * width + col]
// =====================================================================
__global__ void TransposeKernel(
    hipDoubleComplex* d_out,
    const hipDoubleComplex* d_in,
    int width,
    int height
)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        int inIdx = row * width + col;
        int outIdx = col * height + row;
        d_out[outIdx] = d_in[inIdx];
    }
}

// =====================================================================
// 3) Host function to do a batched 1D FFT for either row or column pass
//    length  = number of samples in one row transform
//    batches = how many rows
//    forward = true => sign=-1, else sign=+1 and scale by 1/length
// =====================================================================
static void BatchFFT_1D(
    hipDoubleComplex* d_data,
    int length,
    int batches,
    bool forward
)
{
    // 1) bit-reversal
    // calculates the number of bits needed for the bit-reversal based on the length
    // mathematical equivalent = ceil(log2(length))
    int log2Len = 0;
    while ((1 << log2Len) < length) log2Len++;

    dim3 block(256, 1);
    // ensure enough blocks to cover the length, batches is to run all the row processing in parallel
    dim3 grid((length + block.x - 1) / block.x, batches);

    BitReverseRowKernel << <grid, block >> > (d_data, length, batches, log2Len);
    CHECK_CUDA_ERR("BitReverseRowKernel");

    // 2) iterative passes
    int sign = (forward ? -1 : +1);

    for (int halfSize = 1; halfSize < length; halfSize *= 2) {
        int totalButterflies = length / 2;
        // ensure enough blocks to cover totalButterflies
        dim3 gridBfly((totalButterflies + block.x - 1) / block.x, batches);
        CooleyTukeyRowKernel << <gridBfly, block >> > (d_data, length, batches, halfSize, sign);
        CHECK_CUDA_ERR("CooleyTukeyRowKernel");
    }

    // 3) scale if inverse
    if (!forward) {
        // ensure enough blocks to cover the length
        dim3 gridScale((length + block.x - 1) / block.x, batches);
        double scaleFactor = 1.0 / (double)length;
        ScaleRowKernel << <gridScale, block >> > (d_data, length, batches, scaleFactor);
        CHECK_CUDA_ERR("ScaleRowKernel");
    }
}

// =====================================================================
// 4) Full 2D FFT (rectangular: width x height) in place on d_data
//    forward=true => forward FFT
//    forward=false => inverse FFT
//
//    Step-by-step:
//      - Row pass:   length=width,  batches=height
//      - Transpose:  out = size(width*height)
//      - Column pass: length=height, batches=width
//      - Transpose:  back
//
//    If inverse, each 1D pass scales by (1/length), so total = 1/(width*height).
// =====================================================================
static void FFT2D_CUDA(
    hipDoubleComplex* d_data,
    int width,
    int height,
    bool forward
)
{
    // 1) Row-wise transform: length=width, batches=height
    BatchFFT_1D(d_data, width, height, forward);

    // 2) Transpose (height x width) -> (width x height)
    hipDoubleComplex* d_tmp = nullptr;
    hipMalloc(&d_tmp, width * height * sizeof(hipDoubleComplex));

    dim3 block(16, 16);
    dim3 grid(
        (width + block.x - 1) / block.x,
        (height + block.y - 1) / block.y
    );

    TransposeKernel << <grid, block >> > (d_tmp, d_data, width, height);
    CHECK_CUDA_ERR("TransposeKernel #1");

    // 3) Now d_tmp has shape (width x height) in row-major
    //    So "rows" = width, "row length" = height for the next pass
    BatchFFT_1D(d_tmp, height, width, forward);

    // 4) Transpose back (width x height) -> (height x width)
    //    which is effectively your original row-major shape
    //    i.e., d_data again
    //    We just swap the width/height in the transpose call
    {
        dim3 grid2(
            (height + block.x - 1) / block.x,
            (width + block.y - 1) / block.y
        );
        TransposeKernel << <grid2, block >> > (d_data, d_tmp, height, width);
        CHECK_CUDA_ERR("TransposeKernel #2");
    }

    hipFree(d_tmp);
}

// =====================================================================
// 5) Flatten / unflatten host arrays
// =====================================================================
static void flattenHostArray(
    hipDoubleComplex** input2D,
    hipDoubleComplex* output1D,
    int width,
    int height
)
{
    for (int r = 0; r < height; r++) {
        std::memcpy(
            &output1D[r * width],
            input2D[r],
            width * sizeof(hipDoubleComplex)
        );
    }
}

static void unflattenHostArray(
    const hipDoubleComplex* input1D,
    hipDoubleComplex** output2D,
    int width,
    int height
)
{
    for (int r = 0; r < height; r++) {
        std::memcpy(
            output2D[r],
            &input1D[r * width],
            width * sizeof(hipDoubleComplex)
        );
    }
}

// =====================================================================
// 6) Public function: Forward 2D FFT (rectangular)
// =====================================================================
hipDoubleComplex** FFT2DParallel(uint8_t* inputImage, int width, int height)
{
    hipDoubleComplex** complex_image = storeUint8ToCuComplex2D(inputImage, width, height);

    // Allocate a new host 2D array for the result
    hipDoubleComplex** fft_result = new hipDoubleComplex * [height];
    for (int i = 0; i < height; i++) {
        fft_result[i] = new hipDoubleComplex[width];
    }

    // Flatten input
    hipDoubleComplex* h_temp = new hipDoubleComplex[width * height];
    flattenHostArray(complex_image, h_temp, width, height);

    // GPU alloc
    hipDoubleComplex* d_data = nullptr;
    hipMalloc(&d_data, width * height * sizeof(hipDoubleComplex));

    // Copy to device
    hipMemcpy(d_data, h_temp, width * height * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // Forward 2D FFT
    FFT2D_CUDA(d_data, width, height, true /* forward */);

    // Copy back
    hipMemcpy(h_temp, d_data, width * height * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Unflatten
    unflattenHostArray(h_temp, fft_result, width, height);

    // Cleanup
    delete[] h_temp;
    hipFree(d_data);

    return fft_result;
}

// =====================================================================
// 7) Public function: Inverse 2D FFT (rectangular)
// =====================================================================
uint8_t* IFFT2DParallel(hipDoubleComplex** freqData, int width, int height)
{
    hipDoubleComplex** spatial_result = new hipDoubleComplex * [height];
    for (int i = 0; i < height; i++) {
        spatial_result[i] = new hipDoubleComplex[width];
    }

    // Flatten
    hipDoubleComplex* h_temp = new hipDoubleComplex[width * height];
    flattenHostArray(freqData, h_temp, width, height);

    // GPU alloc
    hipDoubleComplex* d_data = nullptr;
    hipMalloc(&d_data, width * height * sizeof(hipDoubleComplex));
    hipMemcpy(d_data, h_temp, width * height * sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

    // Inverse 2D FFT
    FFT2D_CUDA(d_data, width, height, false /* inverse */);

    // Copy back
    hipMemcpy(h_temp, d_data, width * height * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);

    // Unflatten
    unflattenHostArray(h_temp, spatial_result, width, height);

    // Cleanup
    delete[] h_temp;
    hipFree(d_data);

    return storeCuComplex2DToUint8(spatial_result, width, height);
}

// =====================================================================
// 8) Example test
//    We'll test with a non-square array, e.g. width=6, height=4
// =====================================================================
//static void printComplex2D(
//    hipDoubleComplex** arr,
//    int w, int h,
//    const char* msg,
//    bool showImag = false
//)
//{
//    std::cout << msg << ":\n";
//    for (int r = 0; r < h; r++) {
//        for (int c = 0; c < w; c++) {
//            double re = hipCreal(arr[r][c]);
//            double im = hipCimag(arr[r][c]);
//            if (!showImag) {
//                // Print real part only (to see if we got back original data)
//                std::cout << re << " ";
//            }
//            else {
//                // Full complex
//                std::cout << "(" << re << "," << im << ") ";
//            }
//        }
//        std::cout << "\n";
//    }
//    std::cout << std::endl;
//}
//
//int main()
//{
//    const int width = 4;
//    const int height = 4;
//
//    uint8_t image[width * height] = {
//        1,2,3,4,
//        5,6,7,8,
//        9,10,11,12,
//        13,14,15,16,
//    };
//
//    // Output initial image
//    cout << "Initial Greyscale Image:" << endl;
//    for (int i = 0; i < height; ++i) {
//        for (int j = 0; j < width; ++j) {
//            cout << static_cast<int>(image[i * width + j]) << " ";
//        }
//        cout << endl;
//    }
//
//    // 1) Forward FFT
//    hipDoubleComplex** freq = FFT2DParallel(image, width, height);
//    printComplex2D(freq, width, height, "Frequency domain (full complex)", true);
//
//    // 2) Inverse FFT
//    uint8_t* recon = IFFT2DParallel(freq, width, height);
//    cout << "Reconstructed (real parts)" << endl;
//    for (int i = 0; i < height; ++i) {
//        for (int j = 0; j < width; ++j) {
//            cout << static_cast<int>(recon[i * width + j]) << " ";
//        }
//        cout << endl;
//    }
//
//    // Cleanup
//    for (int r = 0; r < height; r++) {
//        delete[] freq[r];
//    }
//    delete[] freq;
//
//    return 0;
//}
